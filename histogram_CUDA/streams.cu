#include "hip/hip_runtime.h"
/*
CUDA - generation of array of N elements and calculates even and odd numbers occurence - with streams
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG 0
__host__
void errorexit(const char *s) {
    printf("\n%s\n",s); 
    exit(EXIT_FAILURE);   
}

__host__ 
void generate(int *matrix, int matrixSize) {
  srand(time(NULL));
  for(int i=0; i<matrixSize; i++) {
    matrix[i] = rand()%1000;
  }
}

__global__ 
void calculation(int *matrix, int *even, int *odd, int matrixSize, int streamChunk, int streamId) {
    int my_index=blockIdx.x*blockDim.x+threadIdx.x+streamId*streamChunk;

    if(my_index < matrixSize) {
      if(matrix[my_index] % 2) {
        atomicAdd(odd, 1);
      } else {
        atomicAdd(even, 1);
      }
    } 
}

int main(int argc,char **argv) {

  ///define number of streams
  int numberOfStreams = 4;
  
  //define array size and allocate memory on host
  int matrixSize=100000;
  int *hMatrix=(int*)malloc(matrixSize*sizeof(int));

  //get number of chunks to operate per stream
  int streamChunk = matrixSize/numberOfStreams;

  printf("Stream chunk is %d \n", streamChunk);
 
  //define kernel size per stream
  int threadsinblock=1000;
  int blocksingrid=1+((streamChunk-1)/threadsinblock); 

  printf("blocksingrid is %d \n", blocksingrid);
  

  
  //allocate memory for odd and even numbers counters - host
  int *hEven=(int*)malloc(sizeof(int));
  int *hOdd=(int*)malloc(sizeof(int));


  //create streams
  hipStream_t streams[numberOfStreams];
  for(int i=0;i<numberOfStreams;i++) {
      if (hipSuccess!=hipStreamCreate(&streams[i]))
           errorexit("Error creating stream");
    }

  //allocate memory for odd and even numbers counters and array on device and for array on host with hipHostMalloc
  int *dEven=NULL;
  int *dOdd=NULL;
  int *dMatrix=NULL;

  if (hipSuccess!=hipHostMalloc((void **) &hMatrix, matrixSize*sizeof(int)))
      errorexit("Error allocating memory on the CPU");

  //generate random numbers
  generate(hMatrix, matrixSize);

  if(DEBUG) {
    printf("Generated numbers: \n");
    for(int i=0; i<matrixSize; i++) {
      printf("%d ", hMatrix[i]);
    }
    printf("\n");
  }


  if (hipSuccess!=hipMalloc((void **)&dEven,sizeof(int)))
      errorexit("Error allocating memory on the GPU");

  if (hipSuccess!=hipMalloc((void **)&dOdd,sizeof(int)))
      errorexit("Error allocating memory on the GPU");
  
  if (hipSuccess!=hipMalloc((void **)&dMatrix,matrixSize*sizeof(int)))
      errorexit("Error allocating memory on the GPU");

  //initialize allocated counters with 0
  if (hipSuccess!=hipMemset(dEven,0, sizeof(int)))
      errorexit("Error initializing memory on the GPU");

  if(hipSuccess!=hipMemset(dOdd,0, sizeof(int)))
      errorexit("Error initializing memory on the GPU");

  //execute operation in each stream - copy chunk of data and run calculations
  for(int i=0; i<numberOfStreams; i++) {
    hipMemcpyAsync(&dMatrix[streamChunk*i],&hMatrix[streamChunk*i],streamChunk*sizeof(int),hipMemcpyHostToDevice, streams[i]);      
    calculation<<<blocksingrid, threadsinblock, threadsinblock*sizeof(double), streams[i]>>>(dMatrix, dEven, dOdd, matrixSize, streamChunk, i);
  }

  hipDeviceSynchronize();

  //copy results from GPU
  if (hipSuccess!=hipMemcpy(hEven, dEven, sizeof(int),hipMemcpyDeviceToHost))
     errorexit("Error copying results");

  if (hipSuccess!=hipMemcpy(hOdd, dOdd, sizeof(int),hipMemcpyDeviceToHost))
     errorexit("Error copying results");
  
    printf("Found %d even numbers \n", *hEven);
    printf("Found %d odd numbers \n", *hOdd);
    printf("Found %d total numbers \n", *hEven + *hOdd);

//Free memory and destroy streams
    for(int i=0;i<numberOfStreams;i++) {
      if (hipSuccess!=hipStreamDestroy(*(streams+i)))
         errorexit("Error creating stream");
    }

  free(hOdd);
  free(hEven);
  
  if (hipSuccess!=hipHostFree(hMatrix))
     errorexit("Error when deallocating space on the CPU");
  if (hipSuccess!=hipFree(dEven))
    errorexit("Error when deallocating space on the GPU");
  if (hipSuccess!=hipFree(dOdd))
    errorexit("Error when deallocating space on the GPU");
  if (hipSuccess!=hipFree(dMatrix))
    errorexit("Error when deallocating space on the GPU");
}
