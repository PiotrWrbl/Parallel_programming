#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG 0
#define MAX 1000

__host__
void errorexit(const char *s) {
    printf("\n%s\n",s); 
    exit(EXIT_FAILURE);   
}

__host__ 
void generate(int *matrix, int matrixSize) {
  srand(time(NULL));
  for(int i=0; i<matrixSize; i++) {
    matrix[i] = rand()%1000;
  }
}

__global__ 
void calculation(int *matrix, int *histogram,int matrixSize, int streamChunk, int streamId) {
    int my_index=blockIdx.x*blockDim.x+threadIdx.x+streamId*streamChunk;

    if (my_index < matrixSize) {
        atomicAdd(&histogram[matrix[my_index]], 1); // Increment the count of the corresponding number in the histogram
    }
}

int main(int argc,char **argv) {

  ///define number of streams
  int numberOfStreams = 4;
  
  //define array size and allocate memory on host
  int matrixSize=10240000;
  int *hMatrix=NULL;

  //get number of chunks to operate per stream
  int streamChunk = matrixSize/numberOfStreams;

  printf("Stream chunk is %d \n", streamChunk);
 
  //define kernel size per stream
  int threadsinblock=1000;
  int blocksingrid=1+((streamChunk-1)/threadsinblock); 

  printf("blocksingrid is %d \n", blocksingrid);


  // Allocate memory for histogram - host
  int *hHistogram = (int *)malloc(MAX * sizeof(int));

  //create streams
  hipStream_t streams[numberOfStreams];
  for(int i=0;i<numberOfStreams;i++) {
      if (hipSuccess!=hipStreamCreate(&streams[i]))
           errorexit("Error creating stream");
    }

  int *dHistogram = NULL;
  int *dMatrix = NULL;

  if (hipSuccess!=hipHostMalloc((void **) &hMatrix, matrixSize*sizeof(int)))
    errorexit("Error allocating memory on the CPU");

  //generate random numbers
  generate(hMatrix, matrixSize);

  if(DEBUG) {
    printf("Generated numbers: \n");
    for(int i=0; i<matrixSize; i++) {
      printf("%d ", hMatrix[i]);
    }
    printf("\n");
  }


  if (hipSuccess != hipMalloc((void **)&dHistogram, MAX * sizeof(int)))
      errorexit("Error allocating memory on the GPU");

  if (hipSuccess != hipMalloc((void **)&dMatrix, matrixSize * sizeof(int)))
      errorexit("Error allocating memory on the GPU");

  //initialize allocated counter with 0
  if (hipSuccess!=hipMemset(dHistogram,0, MAX * sizeof(int)))
      errorexit("Error initializing memory on the GPU");

  //execute operation in each stream - copy chunk of data and run calculations
  for(int i=0; i<numberOfStreams; i++) {
    hipMemcpyAsync(&dMatrix[streamChunk*i],&hMatrix[streamChunk*i],streamChunk*sizeof(int),hipMemcpyHostToDevice, streams[i]);      
    calculation<<<blocksingrid, threadsinblock, threadsinblock*sizeof(double), streams[i]>>>(dMatrix, dHistogram, matrixSize, streamChunk, i);
  }

  hipDeviceSynchronize();

  //copy results from GPU
  if (hipSuccess != hipMemcpy(hHistogram, dHistogram, MAX * sizeof(int), hipMemcpyDeviceToHost))
      errorexit("Error copying results");
  
    // Display histogram results
  printf("Histogram of occurrences for numbers in range <1, %d>:\n", MAX);
  for (int i = 0; i < MAX; i++) {
      printf("%d: %d times\n", i + 1, hHistogram[i]);
  }

//Free memory and destroy streams
    for(int i=0;i<numberOfStreams;i++) {
      if (hipSuccess!=hipStreamDestroy(streams[i]))
         errorexit("Error creating stream");
    }

  // Free memory
  free(hHistogram);
  
  if (hipSuccess != hipFree(dHistogram))
      errorexit("Error when deallocating space on the GPU");
  if (hipSuccess != hipFree(dMatrix))
      errorexit("Error when deallocating space on the GPU");
  if (hipSuccess != hipHostFree(hMatrix))
      errorexit("Error when deallocating space on the CPU");
}
