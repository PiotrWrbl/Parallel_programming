#include "hip/hip_runtime.h"
/*
CUDA - generation of array of N elements and calculates even and odd numbers occurence - no streams
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG 0
__host__
void errorexit(const char *s) {
		printf("\n%s\n",s); 
		exit(EXIT_FAILURE);   
}

__host__ 
void generate(int *matrix, int matrixSize) {
	srand(time(NULL));
	for(int i=0; i<matrixSize; i++) {
		matrix[i] = rand()%1000;
	}
}

__global__ 
void calculation(int *matrix, int *even, int *odd, int matrixSize) {
		int my_index=blockIdx.x*blockDim.x+threadIdx.x;
		if(my_index < matrixSize) {
			if(matrix[my_index] % 2) {
				atomicAdd(odd, 1);
			} else {
				atomicAdd(even, 1);
			}
		} 
}

int main(int argc,char **argv) {

	//define array size and allocate memory on host
	int matrixSize=100000;
	int *hMatrix=(int*)malloc(matrixSize*sizeof(int));
	
	//generate random numbers
	generate(hMatrix, matrixSize);

	if(DEBUG) {
		printf("Generated numbers: \n");
		for(int i=0; i<matrixSize; i++) {
			printf("%d ", hMatrix[i]);
		}
		printf("\n");
	}

	//allocate memory for odd and even numbers counters - host
	int *hEven=(int*)malloc(sizeof(int));
	int *hOdd=(int*)malloc(sizeof(int));

	//allocate memory for odd and even numbers counters and array - device
	int *dEven=NULL;
	int *dOdd=NULL;
	int *dMatrix=NULL;

	if (hipSuccess!=hipMalloc((void **)&dEven,sizeof(int)))
			errorexit("Error allocating memory on the GPU");

	if (hipSuccess!=hipMalloc((void **)&dOdd,sizeof(int)))
			errorexit("Error allocating memory on the GPU");
	
	if (hipSuccess!=hipMalloc((void **)&dMatrix,matrixSize*sizeof(int)))
			errorexit("Error allocating memory on the GPU");

	//initialize allocated counters with 0
	if (hipSuccess!=hipMemset(dEven,0, sizeof(int)))
			errorexit("Error initializing memory on the GPU");

	if(hipSuccess!=hipMemset(dOdd,0, sizeof(int)))
			errorexit("Error initializing memory on the GPU");

	//copy array to device
	if (hipSuccess!=hipMemcpy(dMatrix,hMatrix,matrixSize*sizeof(int),hipMemcpyHostToDevice))
		 errorexit("Error copying input data to device");

	int threadsinblock=1024;
	int blocksingrid=1+((matrixSize-1)/threadsinblock); 

	//run kernel on GPU 
	calculation<<<blocksingrid, threadsinblock>>>(dMatrix, dEven, dOdd, matrixSize);

	//copy results from GPU
	if (hipSuccess!=hipMemcpy(hEven, dEven, sizeof(int),hipMemcpyDeviceToHost))
		 errorexit("Error copying results");

	if (hipSuccess!=hipMemcpy(hOdd, dOdd, sizeof(int),hipMemcpyDeviceToHost))
		 errorexit("Error copying results");
	
	printf("Found %d even numbers \n", *hEven);
	printf("Found %d odd numbers \n", *hOdd);
	printf("Found %d total numbers \n", *hEven + *hOdd);

	//Free memory
	free(hOdd);
	free(hEven);
	free(hMatrix);
		
	if (hipSuccess!=hipFree(dEven))
		errorexit("Error when deallocating space on the GPU");
	if (hipSuccess!=hipFree(dOdd))
		errorexit("Error when deallocating space on the GPU");
	if (hipSuccess!=hipFree(dMatrix))
		errorexit("Error when deallocating space on the GPU");
}
